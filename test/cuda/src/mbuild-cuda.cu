#include "hip/hip_runtime.h"

/**
 * Kernel function to generate the overlap and Hamiltonian matrices
 */
__global__ void puksham_mbuild_cuda( punity_t *pu, int mdim, int nbp, shape_t *dm, int *nlbase, int **lbase, int *ltg, nuclei_t *nuc, int quadn, double *qpts, double *qwts,
					 long **jap, double **Ap, long **jbp, double **Bp, long cc, long c_spmat_inc,
						int b_use_external_potential, int b_load_overl_mat, int b_load_stiff_mat,
							int b_use_singular, int i_sing_order, int *b_have_stiff_mat, int *b_have_overl_mat )
{
	
}

int main()
{

}

