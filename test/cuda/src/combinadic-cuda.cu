
#include <hip/hip_runtime.h>
/**
 * CUDA device code for combinadic functions
 */

__device__ void partition_init( int *s, int *m, int n )
{
	int i;
	for(i=0;i<n;i++)
		m[i] = 1, s[i] = 1;
}

__device__ int partition_next( int *s, int *m, int n )
{
	/* Update s: 1 1 1 1 -> 2 1 1 1 -> 1 2 1 1 -> 2 2 1 1 -> 3 2 1 1 -> 1 1 2 1 ... */
	int i = 0;
	++s[i];
	while ((i < n - 1) && (s[i] > m[i] + 1))
	{
		s[i] = 1;
		++i;
		++s[i];
	}

	/* If i is has reached n-1 th element, then the last unique partition has been found*/
	if (i == n - 1)
		return 0;

	/* Because all the first i elements are now 1, s[i] (i + 1 th element)
	is the largest. So we update max by copying it to all the first i
	positions in m.*/
	int max = s[i];
	for (i = i - 1; i >= 0; --i)
		m[i] = max;

	return 1;
}

__device__ int factorial( int num_in )
{
	int fct = 1;
	int i;
	if( num_in <= 0 )
		return 1;
	for(i=num_in;i>1;i--)
		fct *= i;
	return fct;
}

__device__ int truncfact( int n_in, int r_in )
{
	int i,prd = 1;
	for(i=0;i<r_in;i++)
		prd *= n_in - i;
	return prd;
}

__device__ int binomial( int n_in, int k_in )
{
	if( n_in < k_in )
		return 0;
	else
		return truncfact( n_in, k_in ) / factorial( k_in );
}

__device__ int combinadic_init( int lim_in, int dim_in, int *ptr_in )
{
	int i;
	if(lim_in<dim_in)
		return -1; // can't make dim_in-combinations of lim_in (< dim_in) objects
	for(i=0;i<dim_in;i++)
		ptr_in[i] = i;
	return 0;
}

__device__ int combinadic_next( int lim_in, int dim_in, int *vec_in )
{
	int i,j;
	for(i=0;i<dim_in;i++)
	{
		if( ( vec_in[i] < lim_in )
			&& ( ( i < dim_in - 1 && ( vec_in[i+1] - vec_in[i] ) > 1 ) || i == dim_in - 1 ) )
		{
			++vec_in[i];
			break;
		}
	}
	for(j=0;j<i;j++)
		vec_in[j] = j;
	return 0;
}

__device__ int combinadic_vector( int idx_in, int lim_in, int dim_in, int *vec_out )
{
	int i,j,idx,tmp;

	if( idx_in < 0 )
		return -1;

	idx = idx_in;
	for(i=dim_in-1;i>=0;i--)
	{
		for(j=i+1;j<=lim_in;j++)
		{
			tmp = binomial( j, i + 1 );
			if( tmp > idx )
				break;
		}
		idx -= binomial( j - 1, i + 1 );
		vec_out[i] = j - 1;
	}
	return 0;
}

__device__ int rcombinadic_vector( int idx_in, int lim_in, int dim_in, int *vec_out )
{
	return combinadic_vector( idx_in, lim_in + dim_in - 1, dim_in - 1, vec_out );
}

__device__ int rcombinadic_occupancy( int lim_in, int dim_in, int *vec_in, int *occ_out )
{
	int i;
	if( dim_in < 1 )
		return -1; /* Failure: Parameter out of acceptable domain */
	if( dim_in > 1 )
	{
		occ_out[0] = vec_in[0];
		for(i=1;i<dim_in-1;i++)
			occ_out[i] = vec_in[i] - vec_in[i-1] - 1;
		occ_out[dim_in-1] = lim_in + dim_in - 1 - vec_in[dim_in-2] - 1; /* minus addtl 'one' adjust difference of 1 to mean zero occupancy */
	}
	else
		occ_out[0] = lim_in;
	return 0;
}

__device__ int polynomial_exponents( int idx_in, int ord_in, int dim_in, int *occ_out )
{
	int *vec = (int*) malloc( ( dim_in-1 ) * sizeof(int) );
	rcombinadic_vector( idx_in, ord_in, dim_in, vec );
	rcombinadic_occupancy( ord_in, dim_in, vec, occ_out );
	free( vec );
	return 0;
}

__device__ int global_polynomial_vector( int idx_in, int dim_in, int *exp_out )
{
	int n,m,r;
	int *cmb = (int*) malloc( ( dim_in - 1 ) * sizeof(int) );

	for(n=0,r=0;;r++)
	{
		m = binomial( r + dim_in - 1, dim_in - 1 );
		if( n + m > idx_in )
			break;
		else
			n += m;
	}

	polynomial_exponents( idx_in - n, r, dim_in, exp_out );

	free( cmb );

	return 0;
}

