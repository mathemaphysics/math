
#include <hip/hip_runtime.h>
/**
 * CUDA device code for stepping through a grid in d dimensions
 */

__device__ int arraynext( long dim_in, long *size_in, long *index_in )
{
	long i;
	for(i=dim_in-1;i>=0;i--)
	{
		if( index_in[i] < size_in[i] )
		{
			++index_in[i];
			return 0;
		}
		else /* Carry to the next spot */
			index_in[i] = 0;
	}
	return -1;
}

