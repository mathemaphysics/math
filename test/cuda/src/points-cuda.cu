#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include "combinadic-cuda.h"
#include "points-cuda.h"
#include "trees-cuda.h"

#define ZERO_THRESHOLD 1e-6

#define CUDA_RAND_MAX 2147483648

typedef struct
{
	int dim;
	int type;
	double *orig;
	double *params;
} shape_t;

__device__ void domain_indicator( double *x, shape_t *sh, int *ret )
{
	int i;
	double sum;

	switch( sh->type )
	{
		case 0:
			*ret = 1;
			for(i=0;i<sh->dim;i++)
			{
				if( x[i] < sh->orig[i] || x[i] > sh->params[0] + sh->orig[i] )
				{
					*ret = 0;
					break;
				}
			}
			break;
		case 1:
			*ret = 1;
			for(i=0;i<sh->dim;i++)
			{
				if( x[i] < sh->orig[i] || x[i] > sh->params[i] + sh->orig[i] )
				{
					*ret = 0;
					break;
				}
			}
			break;
		case 2:
			sum = 0.0;
			for(i=0;i<sh->dim;i++);
				sum += pow( x[i] - sh->orig[i], 2.0 );
			if( sum < sh->params[0] * sh->params[0] )
				*ret = 1;
			else
				*ret = 0;
			break;
		case 3:
			break;
		default:
			*ret = 0;
			break;
	}
}

/**
 * A random number generator for use inside device functions
 */
__device__ unsigned int cuda_rand( unsigned int *m_z, unsigned int *m_w )
{
	*m_z = 36969 * ( (*m_z) & 65535 ) + ( (*m_z) >> 16 );
	*m_w = 18000 * ( (*m_w) & 65535 ) + ( (*m_w) >> 16 );

	return ( ( (*m_z) << 16 ) + (*m_w) ) % CUDA_RAND_MAX;
}

/**
 * Calculate a set of dim_in - 1 axes perpendicular to vec_in
 */
__device__ void local_axes( int dim_in, double *vec_in, double *basis_out )
{
	int i,j,k;
	unsigned int mz,mw;
	double sum;

	sum = 0.0;
	for(i=0;i<dim_in;i++)
		basis_out[i] = vec_in[i], sum += basis_out[i] * basis_out[i];
	sum = sqrt( sum );
	for(i=0;i<dim_in;i++)
		basis_out[i] /= sum;
	mz = 150; mw = 40;
	for(i=1;i<dim_in;i++)
	{
		/* This is a horrible idea */
		for(j=0;j<dim_in;j++)
			basis_out[i*dim_in+j] = 0.5 - ( (double) cuda_rand( &mz, &mw ) / (double) CUDA_RAND_MAX );
		sum = 0.0;
		for(j=0;j<dim_in;j++)
			sum += basis_out[i*dim_in+j] * basis_out[i*dim_in+j];
		sum = sqrt( sum );
		for(j=0;j<dim_in;j++)
			basis_out[i*dim_in+j] /= sum;
		for(j=0;j<i;j++)
		{
			sum = 0.0;
			for(k=0;k<dim_in;k++)
				sum += basis_out[i*dim_in+k] * basis_out[j*dim_in+k];
			for(k=0;k<dim_in;k++)
				basis_out[i*dim_in+k] -= sum * basis_out[j*dim_in+k];
		}
		sum = 0.0;
		for(j=0;j<dim_in;j++)
			sum += basis_out[i*dim_in+j] * basis_out[i*dim_in+j];
		sum = sqrt( sum );
		for(j=0;j<dim_in;j++)
			basis_out[i*dim_in+j] /= sum;
	}
}

/**
 * Calculate a rectangular box about the intersection of two spheres
 * of potentially different radii
 * @param dim_in Dimension in which the spheres live
 * @param ctr1_in Center of the first sphere
 * @param rad1_in Radius of the first sphere
 * @param ctr2_in Center of the second sphere
 * @param rad2_in Radius of the second sphere
 * @param ctr_out Center of the disc containing the intersection
 * @param rad_out Radius of the disc containing the intersection
 * @param qbox_out Contains a local basis covering the intersection of the spheres
 * @return Returns 1 if spheres intersect, 0 if not
 */
__device__ int sphere_intersection( int dim_in, double *ctr1_in, double rad1_in, double *ctr2_in, double rad2_in, double *ctr_out, double *rad_out, double *qbox_out )
{
	int i,j;
	double r,rr,sum;
	double *ax = (double*) malloc( dim_in * sizeof(double) );

	/* Calculate the origin and axis of the cylinder */
	sum = 0.0;
	for(i=0;i<dim_in;i++)
		ax[i] = ctr2_in[i] - ctr1_in[i], sum += ax[i] * ax[i];
	sum = sqrt( sum );
	if( sum > rad1_in + rad2_in )
		return 0;

	/* If circles are identical */
	if( sum < ZERO_THRESHOLD )
	{
		/* Then build a box around the smaller sphere */
		for(i=0;i<dim_in;i++)
			ctr_out[i] = ctr1_in[i];
		*rad_out = ( rad1_in < rad2_in ) ? rad1_in : rad2_in;
		for(i=0;i<dim_in;i++)
			for(j=0;j<dim_in;j++)
				qbox_out[i*dim_in+j] = ( i == j ) ? *rad_out : 0.0;
		return 1;
	}

	/* Otherwise */
	for(i=0;i<dim_in;i++)
		ax[i] /= sum;
	r = rad1_in + rad2_in - sum;
	for(i=0;i<dim_in;i++)
		ctr_out[i] = ctr1_in[i] + ( sum - rad2_in ) * ax[i];
	for(i=0;i<dim_in;i++)
		ax[i] *= r;

	/* Calculate the radius of the cylinder */
	rr = sqrt( fabs( rad1_in * rad1_in - rad2_in * rad2_in ) );
	if( sum < rr )
		*rad_out = ( rad1_in < rad2_in ) ? rad1_in : rad2_in;
	else
		*rad_out = sqrt( ( -sum + rad2_in - rad1_in ) * ( -sum - rad2_in + rad1_in )
			* ( -sum + rad2_in + rad1_in ) * ( sum + rad2_in + rad1_in ) ) / 2.0 / sum;

	/* Generate the local coordinates */
	local_axes( dim_in, ax, qbox_out );
	for(i=0;i<dim_in;i++)
		qbox_out[i] = ax[i];
	for(i=1;i<dim_in;i++)
		for(j=0;j<dim_in;j++)
			qbox_out[i*dim_in+j] *= (*rad_out);

	/* Move ctr_out to the middle of the box */
	for(i=0;i<dim_in;i++)
		qbox_out[i] *= 0.5, ctr_out[i] += qbox_out[i];

	free( ax );

	return 1;
}

/**
 * Generate a quadrature point in the lens of spherical intersection
 */
__device__ void lens_gauss_point( int dim_in,
		double *ctr1_in, double rad1_in,
		double *ctr2_in, double rad2_in,
		double cr_in, double *nqbox_in, long *index_in,
		double *qpts_in, double *qwts_in,
		double *qp_out, double *qw_out )
{
	int i,j;
	double ssum,x1,x2;
	double *dr,*vec,*wec;

	/* Must allocate memory via malloc in __device__ code */
	dr = (double*) malloc( ( dim_in - 1 ) * sizeof(double) );
	vec = (double*) malloc( dim_in * sizeof(double) );
	wec = (double*) malloc( dim_in * sizeof(double) );

	/* Calculate distance between centers */
	ssum = 0.0;
	for(i=0;i<dim_in;i++)
		ssum += pow( ctr2_in[i] - ctr1_in[i], 2.0 );
	ssum = sqrt( ssum );
	
	/* Calculate the limits for each dimension */
	for(i=0;i<dim_in-1;i++)
	{
		dr[i] = cr_in * cr_in;
		for(j=0;j<i;j++)
			dr[i] -= dr[j] * qpts_in[index_in[j]] * dr[j] * qpts_in[index_in[j]];
		dr[i] = sqrt( dr[i] );
		vec[i] = dr[i] * qpts_in[index_in[i]];
	}

	/* Now for the final dimension which is a function of sphere separation */
	x1 = rad1_in * rad1_in;
	for(i=0;i<dim_in-1;i++)
		x1 -= vec[i] * vec[i];
	x1 = sqrt( x1 );
	x2 = rad2_in * rad2_in;
	for(i=0;i<dim_in-1;i++)
		x2 -= vec[i] * vec[i];
	x2 = ssum - sqrt( x2 );

	/* Project the point onto the whole domain */
	for(i=0;i<dim_in;i++)
      		wec[i] = ctr1_in[i]; /* Translate to the origin, ctr1_in */
	for(i=1;i<dim_in;i++)
		for(j=0;j<dim_in;j++)
			wec[j] += vec[i-1] * nqbox_in[i*dim_in+j];

	/* Index along the axis is given by index_in[dim_in-1] */
	for(i=0;i<dim_in;i++)
		qp_out[i] = wec[i] + ( 0.5 * ( x1 + x2 ) + 0.5 * ( x1 - x2 ) * qpts_in[index_in[dim_in-1]] ) * nqbox_in[i];
	*qw_out = qwts_in[index_in[dim_in-1]];
	for(i=1;i<dim_in;i++)
		*qw_out *= qwts_in[index_in[i-1]] * 2.0 * dr[i-1];
	*qw_out *= fabs( x2 - x1 );
}

__device__ void sphere_gauss_point( int dim_in, double *ctr_in, double rad_in, double *nqbox_in, long *index_in, double *qpts_in, double *qwts_in, double *qp_out, double *qw_out )
{
	int i,j;
	double *dr,*vec;

	/* Allocate directly */
	dr = (double*) malloc( dim_in * sizeof(double) );
	vec = (double*) malloc( dim_in * sizeof(double) );

	/* Calculate dimension limits */
	for(i=0;i<dim_in;i++)
	{
		dr[i] = rad_in * rad_in;
		for(j=0;j<i;j++)
			dr[i] -= dr[j] * qpts_in[index_in[j]] * dr[j] * qpts_in[index_in[j]];
		dr[i] = sqrt( dr[i] );
		vec[i] = dr[i] * qpts_in[index_in[i]];
	}

	/* Project the point onto the entire domain by affine transformation */
	for(i=0;i<dim_in;i++)
		qp_out[i] = ctr_in[i];
	for(i=0;i<dim_in;i++)
		for(j=0;j<dim_in;j++)
			qp_out[j] += vec[i] * nqbox_in[i*dim_in+j];
	*qw_out = 1.0;
	for(i=0;i<dim_in;i++)
		(*qw_out) *= 2.0 * dr[i] * qwts_in[index_in[i]];
}

