#include "hip/hip_runtime.h"
/**
 * CUDA device code for combinadic functions
 */

typedef struct
{
	int dim;
	int np;
	double *pts;
	double *dlt;
	double *params;
	double *chg;
} nuclei_t;

__device__ void partition_init( int *s, int *m, int n )
{
	int i;
	for(i=0;i<n;i++)
		m[i] = 1, s[i] = 1;
}

__device__ int partition_next( int *s, int *m, int n )
{
	/* Update s: 1 1 1 1 -> 2 1 1 1 -> 1 2 1 1 -> 2 2 1 1 -> 3 2 1 1 -> 1 1 2 1 ... */
	int i = 0;
	++s[i];
	while ((i < n - 1) && (s[i] > m[i] + 1))
	{
		s[i] = 1;
		++i;
		++s[i];
	}

	/* If i is has reached n-1 th element, then the last unique partition has been found*/
	if (i == n - 1)
		return 0;

	/* Because all the first i elements are now 1, s[i] (i + 1 th element)
	is the largest. So we update max by copying it to all the first i
	positions in m.*/
	int max = s[i];
	for (i = i - 1; i >= 0; --i)
		m[i] = max;

	return 1;
}

__device__ int factorial( int num_in )
{
	int fct = 1;
	int i;
	if( num_in <= 0 )
		return 1;
	for(i=num_in;i>1;i--)
		fct *= i;
	return fct;
}

__device__ int truncfact( int n_in, int r_in )
{
	int i,prd = 1;
	for(i=0;i<r_in;i++)
		prd *= n_in - i;
	return prd;
}

__device__ int binomial( int n_in, int k_in )
{
	if( n_in < k_in )
		return 0;
	else
		return truncfact( n_in, k_in ) / factorial( k_in );
}

__device__ int combinadic_init( int lim_in, int dim_in, int *ptr_in )
{
	int i;
	if(lim_in<dim_in)
		return -1; // can't make dim_in-combinations of lim_in (< dim_in) objects
	for(i=0;i<dim_in;i++)
		ptr_in[i] = i;
	return 0;
}

__device__ int combinadic_next( int lim_in, int dim_in, int *vec_in )
{
	int i,j;
	for(i=0;i<dim_in;i++)
	{
		if( ( vec_in[i] < lim_in )
			&& ( ( i < dim_in - 1 && ( vec_in[i+1] - vec_in[i] ) > 1 ) || i == dim_in - 1 ) )
		{
			++vec_in[i];
			break;
		}
	}
	for(j=0;j<i;j++)
		vec_in[j] = j;
	return 0;
}

__device__ int combinadic_vector( int idx_in, int lim_in, int dim_in, int *vec_out )
{
	int i,j,idx,tmp;

	if( idx_in < 0 )
		return -1;

	idx = idx_in;
	for(i=dim_in-1;i>=0;i--)
	{
		for(j=i+1;j<=lim_in;j++)
		{
			tmp = binomial( j, i + 1 );
			if( tmp > idx )
				break;
		}
		idx -= binomial( j - 1, i + 1 );
		vec_out[i] = j - 1;
	}
	return 0;
}

__device__ int rcombinadic_vector( int idx_in, int lim_in, int dim_in, int *vec_out )
{
	return combinadic_vector( idx_in, lim_in + dim_in - 1, dim_in - 1, vec_out );
}

__device__ int rcombinadic_occupancy( int lim_in, int dim_in, int *vec_in, int *occ_out )
{
	int i;
	if( dim_in < 1 )
		return -1; /* Failure: Parameter out of acceptable domain */
	if( dim_in > 1 )
	{
		occ_out[0] = vec_in[0];
		for(i=1;i<dim_in-1;i++)
			occ_out[i] = vec_in[i] - vec_in[i-1] - 1;
		occ_out[dim_in-1] = lim_in + dim_in - 1 - vec_in[dim_in-2] - 1; /* minus addtl 'one' adjust difference of 1 to mean zero occupancy */
	}
	else
		occ_out[0] = lim_in;
	return 0;
}

__device__ int polynomial_exponents( int idx_in, int ord_in, int dim_in, int *occ_out )
{
	int *vec = (int*) malloc( ( dim_in-1 ) * sizeof(int) );
	rcombinadic_vector( idx_in, ord_in, dim_in, vec );
	rcombinadic_occupancy( ord_in, dim_in, vec, occ_out );
	free( vec );
	return 0;
}

__device__ int global_polynomial_vector( int idx_in, int dim_in, int *exp_out )
{
	int n,m,r;
	int *cmb = (int*) malloc( ( dim_in - 1 ) * sizeof(int) );

	for(n=0,r=0;;r++)
	{
		m = binomial( r + dim_in - 1, dim_in - 1 );
		if( n + m > idx_in )
			break;
		else
			n += m;
	}

	polynomial_exponents( idx_in - n, r, dim_in, exp_out );

	free( cmb );

	return 0;
}

/**
 * CUDA device code for stepping through a grid in d dimensions
 */

__device__ int arraynext( long dim_in, long *size_in, long *index_in )
{
	long i;
	for(i=dim_in-1;i>=0;i--)
	{
		if( index_in[i] < size_in[i] )
		{
			++index_in[i];
			return 0;
		}
		else /* Carry to the next spot */
			index_in[i] = 0;
	}
	return -1;
}

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define ZERO_THRESHOLD 1e-6

#define CUDA_RAND_MAX 2147483648

typedef struct
{
	int dim;
	int type;
	double *orig;
	double *params;
} shape_t;

__device__ void domain_indicator( double *x, shape_t *sh, int *ret )
{
	int i;
	double sum;

	switch( sh->type )
	{
		case 0:
			*ret = 1;
			for(i=0;i<sh->dim;i++)
			{
				if( x[i] < sh->orig[i] || x[i] > sh->params[0] + sh->orig[i] )
				{
					*ret = 0;
					break;
				}
			}
			break;
		case 1:
			*ret = 1;
			for(i=0;i<sh->dim;i++)
			{
				if( x[i] < sh->orig[i] || x[i] > sh->params[i] + sh->orig[i] )
				{
					*ret = 0;
					break;
				}
			}
			break;
		case 2:
			sum = 0.0;
			for(i=0;i<sh->dim;i++);
				sum += pow( x[i] - sh->orig[i], 2.0 );
			if( sum < sh->params[0] * sh->params[0] )
				*ret = 1;
			else
				*ret = 0;
			break;
		case 3:
			break;
		default:
			*ret = 0;
			break;
	}
}

/**
 * A random number generator for use inside device functions
 */
__device__ unsigned int cuda_rand( unsigned int *m_z, unsigned int *m_w )
{
	*m_z = 36969 * ( (*m_z) & 65535 ) + ( (*m_z) >> 16 );
	*m_w = 18000 * ( (*m_w) & 65535 ) + ( (*m_w) >> 16 );

	return ( ( (*m_z) << 16 ) + (*m_w) ) % CUDA_RAND_MAX;
}

/**
 * Calculate a set of dim_in - 1 axes perpendicular to vec_in
 */
__device__ void local_axes( int dim_in, double *vec_in, double *basis_out )
{
	int i,j,k;
	unsigned int mz,mw;
	double sum;

	sum = 0.0;
	for(i=0;i<dim_in;i++)
		basis_out[i] = vec_in[i], sum += basis_out[i] * basis_out[i];
	sum = sqrt( sum );
	for(i=0;i<dim_in;i++)
		basis_out[i] /= sum;
	mz = 150; mw = 40;
	for(i=1;i<dim_in;i++)
	{
		/* This is a horrible idea */
		for(j=0;j<dim_in;j++)
			basis_out[i*dim_in+j] = 0.5 - ( (double) cuda_rand( &mz, &mw ) / (double) CUDA_RAND_MAX );
		sum = 0.0;
		for(j=0;j<dim_in;j++)
			sum += basis_out[i*dim_in+j] * basis_out[i*dim_in+j];
		sum = sqrt( sum );
		for(j=0;j<dim_in;j++)
			basis_out[i*dim_in+j] /= sum;
		for(j=0;j<i;j++)
		{
			sum = 0.0;
			for(k=0;k<dim_in;k++)
				sum += basis_out[i*dim_in+k] * basis_out[j*dim_in+k];
			for(k=0;k<dim_in;k++)
				basis_out[i*dim_in+k] -= sum * basis_out[j*dim_in+k];
		}
		sum = 0.0;
		for(j=0;j<dim_in;j++)
			sum += basis_out[i*dim_in+j] * basis_out[i*dim_in+j];
		sum = sqrt( sum );
		for(j=0;j<dim_in;j++)
			basis_out[i*dim_in+j] /= sum;
	}
}

/**
 * Calculate a rectangular box about the intersection of two spheres
 * of potentially different radii
 * @param dim_in Dimension in which the spheres live
 * @param ctr1_in Center of the first sphere
 * @param rad1_in Radius of the first sphere
 * @param ctr2_in Center of the second sphere
 * @param rad2_in Radius of the second sphere
 * @param ctr_out Center of the disc containing the intersection
 * @param rad_out Radius of the disc containing the intersection
 * @param qbox_out Contains a local basis covering the intersection of the spheres
 * @return Returns 1 if spheres intersect, 0 if not
 */
__device__ int sphere_intersection( int dim_in, double *ctr1_in, double rad1_in, double *ctr2_in, double rad2_in, double *ctr_out, double *rad_out, double *qbox_out )
{
	int i,j;
	double r,rr,sum;
	double *ax = (double*) malloc( dim_in * sizeof(double) );

	/* Calculate the origin and axis of the cylinder */
	sum = 0.0;
	for(i=0;i<dim_in;i++)
		ax[i] = ctr2_in[i] - ctr1_in[i], sum += ax[i] * ax[i];
	sum = sqrt( sum );
	if( sum > rad1_in + rad2_in )
		return 0;

	/* If circles are identical */
	if( sum < ZERO_THRESHOLD )
	{
		/* Then build a box around the smaller sphere */
		for(i=0;i<dim_in;i++)
			ctr_out[i] = ctr1_in[i];
		*rad_out = ( rad1_in < rad2_in ) ? rad1_in : rad2_in;
		for(i=0;i<dim_in;i++)
			for(j=0;j<dim_in;j++)
				qbox_out[i*dim_in+j] = ( i == j ) ? *rad_out : 0.0;
		return 1;
	}

	/* Otherwise */
	for(i=0;i<dim_in;i++)
		ax[i] /= sum;
	r = rad1_in + rad2_in - sum;
	for(i=0;i<dim_in;i++)
		ctr_out[i] = ctr1_in[i] + ( sum - rad2_in ) * ax[i];
	for(i=0;i<dim_in;i++)
		ax[i] *= r;

	/* Calculate the radius of the cylinder */
	rr = sqrt( fabs( rad1_in * rad1_in - rad2_in * rad2_in ) );
	if( sum < rr )
		*rad_out = ( rad1_in < rad2_in ) ? rad1_in : rad2_in;
	else
		*rad_out = sqrt( ( -sum + rad2_in - rad1_in ) * ( -sum - rad2_in + rad1_in )
			* ( -sum + rad2_in + rad1_in ) * ( sum + rad2_in + rad1_in ) ) / 2.0 / sum;

	/* Generate the local coordinates */
	local_axes( dim_in, ax, qbox_out );
	for(i=0;i<dim_in;i++)
		qbox_out[i] = ax[i];
	for(i=1;i<dim_in;i++)
		for(j=0;j<dim_in;j++)
			qbox_out[i*dim_in+j] *= (*rad_out);

	/* Move ctr_out to the middle of the box */
	for(i=0;i<dim_in;i++)
		qbox_out[i] *= 0.5, ctr_out[i] += qbox_out[i];

	free( ax );

	return 1;
}

/**
 * Generate a quadrature point in the lens of spherical intersection
 */
__device__ void lens_gauss_point( int dim_in,
		double *ctr1_in, double rad1_in,
		double *ctr2_in, double rad2_in,
		double cr_in, double *nqbox_in, long *index_in,
		double *qpts_in, double *qwts_in,
		double *qp_out, double *qw_out )
{
	int i,j;
	double ssum,x1,x2;
	double *dr,*vec,*wec;

	/* Must allocate memory via malloc in __device__ code */
	dr = (double*) malloc( ( dim_in - 1 ) * sizeof(double) );
	vec = (double*) malloc( dim_in * sizeof(double) );
	wec = (double*) malloc( dim_in * sizeof(double) );

	/* Calculate distance between centers */
	ssum = 0.0;
	for(i=0;i<dim_in;i++)
		ssum += pow( ctr2_in[i] - ctr1_in[i], 2.0 );
	ssum = sqrt( ssum );
	
	/* Calculate the limits for each dimension */
	for(i=0;i<dim_in-1;i++)
	{
		dr[i] = cr_in * cr_in;
		for(j=0;j<i;j++)
			dr[i] -= dr[j] * qpts_in[index_in[j]] * dr[j] * qpts_in[index_in[j]];
		dr[i] = sqrt( dr[i] );
		vec[i] = dr[i] * qpts_in[index_in[i]];
	}

	/* Now for the final dimension which is a function of sphere separation */
	x1 = rad1_in * rad1_in;
	for(i=0;i<dim_in-1;i++)
		x1 -= vec[i] * vec[i];
	x1 = sqrt( x1 );
	x2 = rad2_in * rad2_in;
	for(i=0;i<dim_in-1;i++)
		x2 -= vec[i] * vec[i];
	x2 = ssum - sqrt( x2 );

	/* Project the point onto the whole domain */
	for(i=0;i<dim_in;i++)
      		wec[i] = ctr1_in[i]; /* Translate to the origin, ctr1_in */
	for(i=1;i<dim_in;i++)
		for(j=0;j<dim_in;j++)
			wec[j] += vec[i-1] * nqbox_in[i*dim_in+j];

	/* Index along the axis is given by index_in[dim_in-1] */
	for(i=0;i<dim_in;i++)
		qp_out[i] = wec[i] + ( 0.5 * ( x1 + x2 ) + 0.5 * ( x1 - x2 ) * qpts_in[index_in[dim_in-1]] ) * nqbox_in[i];
	*qw_out = qwts_in[index_in[dim_in-1]];
	for(i=1;i<dim_in;i++)
		*qw_out *= qwts_in[index_in[i-1]] * 2.0 * dr[i-1];
	*qw_out *= fabs( x2 - x1 );
}

__device__ void sphere_gauss_point( int dim_in, double *ctr_in, double rad_in, double *nqbox_in, long *index_in, double *qpts_in, double *qwts_in, double *qp_out, double *qw_out )
{
	int i,j;
	double *dr,*vec;

	/* Allocate directly */
	dr = (double*) malloc( dim_in * sizeof(double) );
	vec = (double*) malloc( dim_in * sizeof(double) );

	/* Calculate dimension limits */
	for(i=0;i<dim_in;i++)
	{
		dr[i] = rad_in * rad_in;
		for(j=0;j<i;j++)
			dr[i] -= dr[j] * qpts_in[index_in[j]] * dr[j] * qpts_in[index_in[j]];
		dr[i] = sqrt( dr[i] );
		vec[i] = dr[i] * qpts_in[index_in[i]];
	}

	/* Project the point onto the entire domain by affine transformation */
	for(i=0;i<dim_in;i++)
		qp_out[i] = ctr_in[i];
	for(i=0;i<dim_in;i++)
		for(j=0;j<dim_in;j++)
			qp_out[j] += vec[i] * nqbox_in[i*dim_in+j];
	*qw_out = 1.0;
	for(i=0;i<dim_in;i++)
		(*qw_out) *= 2.0 * dr[i] * qwts_in[index_in[i]];
}

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

#define PUNITY_NEIGHBOR_INC 512
#define PUNITY_NB_INCREMENT 512

typedef struct
{
	/**
	 * Dimension of the point space
	 */
	int dim;

	/**
	 * The number of window functions
	 */
	int npts;

	/**
	 * List of the actual points
	 */
	double *pts;

	/**
	 * List of dilation factors for each point
	 */
	double *dlt;

	/**
	 * Window function pointer
	 */
	double (*wfs)(int,double,double*);

	/**
	 * Pointer to function returning gradient of
	 * the window function
	 */
	double (*wfsd)(int,int,double,double*);

	/**
	 * Maximum value of dilation factor
	 */
	double rmax;

	/**
	 * The kdtree storing the point list for
	 * rapid access
	 */
	void *kdt;

	/**
	 * List of ones and zeros; if bdry[i] = 1
	 * then function i centered at point i is
	 * on the boundary
	 */
	char *bdry;
} punity_t;

__device__ double cubic_window( int dim_in, double a_in, double *x_in )
{
        int i;
        double z = 0.0;
#ifdef PUNITY_NORMALIZE
	double vol = pow( a_in, (double) dim_in );
#else
	double vol = 1.0;
#endif
        for(i=0;i<dim_in;i++)
                z += x_in[i] * x_in[i];
        z = sqrt( z ) / a_in * 2.0;

        if( z > 2.0 )
                return 0.0 / vol;
        if( z > 1.0 )
                return ( 2.0 - z ) * ( 2.0 - z ) * ( 2.0 - z ) / 6.0 / vol;
        if( z >= 0.0 )
                return ( 4.0 - 6.0 * z * z + 3.0 * z * z * z ) / 6.0 / vol;
}

__device__ double cubic_window_deriv( int dim_in, int drv_in, double a_in, double *x_in )
{
	int i,j;
	double sum,prd,z = 0.0;

	/* Initialize polynomial coefficients correctly pre-scaled */
	double cf1[4] = { 4.0 / 3.0, 2.0 * -2.0 / a_in, 4.0 * 1.0 / a_in / a_in, 8.0 * -1.0 / 6.0 / a_in / a_in / a_in };
	double cf2[4] = { 2.0 / 3.0, 2.0 * 0.0 / a_in, 4.0 * -1.0 / a_in / a_in, 8.0 * 1.0 / 2.0 / a_in / a_in / a_in };

	/* Calculate distance from zero */
	for(i=0;i<dim_in;i++)
		z += x_in[i] * x_in[i];
        z = sqrt( z );

#ifdef PUNITY_NORMALIZE
	double vol = pow( a_in, (double) dim_in );
#else
	double vol = 1.0;
#endif

	/* Evaluate the drv_in derivative for each case */
	sum = 0.0;
	if( z / a_in > 1.0 )
		return 0.0 / vol;
	if( z / a_in > 0.5 )
	{
		/* Differentiate each term and evaluate at z */
		for(i=0;i<4;i++) /* Variable i doubles as the order of the current term */
		{
			prd = cf1[i];
			for(j=0;j<drv_in;j++)
				prd *= (double) ( i - j );
			prd *= pow( z, (double) ( i - drv_in ) );
			sum += prd;
		}
		return sum / vol;
	}
	if( z / a_in >= 0.0 )
	{
		/* Differentiate each term and evaluate at z */
                for(i=0;i<4;i++) /* Variable i doubles as the order of the current term */
                {
                        prd = cf2[i];
                        for(j=0;j<drv_in;j++)
                                prd *= (double) ( i - j );
                        prd *= pow( z, (double) ( i - drv_in ) );
                        sum += prd;
                }
		return sum / vol;
	}
}

__device__ double quartic_window( int dim_in, double a_in, double *x_in )
{
	int i;
	double z = 0.0;
	for(i=0;i<dim_in;i++)
		z += x_in[i] * x_in[i];
	z = sqrt( z ) / a_in;

#ifdef PUNITY_NORMALIZE
	double vol = pow( a_in, (double) dim_in );
#else
	double vol = 1.0;
#endif

	if( z > 1.0 )
		return 0.0 / vol;
	else
		return ( 1.0 - 6.0 * z * z + 8.0 * z * z * z - 3.0 * z * z * z * z ) / vol;
}

__device__ double quartic_window_deriv( int dim_in, int drv_in, double a_in, double *x_in )
{
	int i;

	
}

/**
 * Evaluate the window function; IMPORTANT: This function shifts the window
 * functions and places it at the points stored in obj_in->pts + idx_in points;
 * obj_in->wfs and obj_in->wfsd do not translation, but they do dilate
 * @param obj_in PU object
 * @param idx_in Window index
 * @param x_in Point at which to evaluate
 */
__device__ double punity_window_evaluate( punity_t *obj_in, int idx_in, double *x_in )
{
	int i;
	double y;
	double *x = (double*) malloc( obj_in->dim * sizeof(double) );

	for(i=0;i<obj_in->dim;i++)
		x[i] = x_in[i] - obj_in->pts[idx_in*obj_in->dim+i];
	y = obj_in->wfs( obj_in->dim, obj_in->dlt[idx_in], x );
	free( x );
	return y;
}

/**
 * Evaluate the window function placing a singularity at the
 * origin of the function for generating a partition of unity
 * with the Kronecker delta property; IMPORTANT: This function
 * also translates the origin to the point in obj_in->pts + idx_in
 * @param obj_in PU object
 * @param idx_in Window index
 * @param x_in Point at which to evaluate
 * @param exp_in Exponent to define the singularity
 */
__device__ double punity_window_evaluate_delta( punity_t *obj_in, int idx_in, double *x_in, int exp_in )
{
	int i;
	double r = 0.0;

	for(i=0;i<obj_in->dim;i++)
		r += pow( x_in[i] - obj_in->pts[idx_in*obj_in->dim+i], 2.0 );
	r = sqrt( r );

	return punity_window_evaluate( obj_in, idx_in, x_in ) / pow( r / obj_in->dlt[idx_in], (double) exp_in );
}

/**
 * Initialize the internal data structures for storage of points
 * and particle radii.
 * @param obj_in The partition of unity data structure
 * @param dim_in The dimension of the point space
 * @param npts_in Number of points to use to generate the partition
 * @param pts_in Set of points to use to generate the partition
 * @param dlt_in Set of dilation or scale factors or radii of particles
 * @param wfs_in Window function to use to generate the partition
 * @param wfsd_in Function returning specific derivatives of the window
 * @return Returns 0 if no error, -1 if memory issues
 */
__device__ int punity_init( punity_t *obj_in, int dim_in, int npts_in, double *pts_in, double *dlt_in, double (*wfs_in)(int,double,double*), double (*wfsd_in)(int,int,double,double*) )
{
	int i;

	/* Set all the basic dimensional information */
	obj_in->dim = dim_in;
	obj_in->npts = npts_in;
	obj_in->pts = (double*) malloc( dim_in * npts_in * sizeof(double) );
	obj_in->dlt = (double*) malloc( npts_in * sizeof(double) );
	if( obj_in->pts == NULL || obj_in->dlt == NULL )
		return -1;

	/* Copy the point data into the structure */
	for(i=0;i<dim_in*npts_in;i++)
		obj_in->pts[i] = pts_in[i];
	for(i=0;i<npts_in;i++)
		obj_in->dlt[i] = dlt_in[i];

	/* Set the function pointers */
	obj_in->wfs = wfs_in;
	obj_in->wfsd = wfsd_in;

	/* Calculate the maximum dilation factor in the system */
	for(i=0;i<npts_in;i++)
		if( i == 0 || dlt_in[i] > obj_in->rmax )
			obj_in->rmax = dlt_in[i];

	/* Initialize all points to internal points; set boundary points to 1 later */
	obj_in->bdry = (char*) malloc( npts_in * sizeof(char) );
	for(i=0;i<npts_in;i++)
		obj_in->bdry[i] = 0;

	return 0;
}

/**
 * Clean up memory allocated for punity_t
 * @param obj_in PU object to clean up
 */
__device__ int punity_free( punity_t *obj_in )
{
	free( obj_in->pts );
	free( obj_in->dlt );
	free( obj_in->bdry );
}

/**
 * Evaluate the particle function idx_in at point x_in.
 * @param obj_in Partition of unity object
 * @param idx_in Index of the function to evaluate
 * @param x_in Position at which to evaluate the function
 * @return Value of function idx_in at point x_in
 */
__device__ double punity_evaluate( punity_t *obj_in, int idx_in, double *x_in )
{
	int i,j;
	double res,sum,y;
	double *vec = (double*) malloc( obj_in->dim * sizeof(double) );

	/* If an index is given outside the range of number of points then return zero */
	if( idx_in > obj_in->npts - 1 )
		return 0.0;

	/* Don't bother with the denominator if the numerator is zero */
	sum = 0.0;
	for(i=0;i<obj_in->dim;i++)
		sum += pow( obj_in->pts[idx_in*obj_in->dim+i] - x_in[i], 2.0 );
	if( sum > obj_in->dlt[idx_in] * obj_in->dlt[idx_in] )
		return 0.0;

	/* NOTE: Change this to punity_neighbors_fast() */
	sum = 0.0;
	for(i=0;i<obj_in->npts;i++)
	{
		res = 0.0;
		for(j=0;j<obj_in->dim;j++)
			vec[j] = x_in[j] - obj_in->pts[i*obj_in->dim+j], res += vec[j] * vec[j];
		if( res < obj_in->dlt[i] * obj_in->dlt[i] )
			sum += obj_in->wfs( obj_in->dim, obj_in->dlt[i], vec );
	}
	for(i=0;i<obj_in->dim;i++)
		vec[i] = x_in[i] - obj_in->pts[idx_in*(obj_in->dim)+i];
	y = obj_in->wfs( obj_in->dim, obj_in->dlt[idx_in], vec ) / sum;
	free( vec );
	return y;
}

/**
 * Evaluate the partition of unity with singularities if needed; this function
 * reads the values of obj_in->bdry for all functions in the domain in order
 * to decide which windows to evaluate with singularities; this is mainly used
 * to make boundary conditions easier to implement, but it can be used to implement
 * interpolating partitions of unity in general (although there may be degradation
 * of interpolants); in the future, make the exponent a function of the function
 * index, i.e. store as obj_in->exp[i]
 * @param obj_in The PU object
 * @param idx_in Index of the function to evaluate
 * @param x_in Point at which to evaluate the functions
 * @param exp_in Exponent to use for singularity evaluation
 */
__device__ double punity_evaluate_delta( punity_t *obj_in, int idx_in, double *x_in, int exp_in )
{
	int i,j;
	double res,sum,y;
	double *vec = (double*) malloc( obj_in->dim * sizeof(double) );

	/* If an index is given outside the range of number of points then return zero */
	if( idx_in > obj_in->npts - 1 )
		return 0.0;

	/* Don't bother with the denominator if the numerator is zero */
	sum = 0.0;
	for(i=0;i<obj_in->dim;i++)
		sum += pow( obj_in->pts[idx_in*obj_in->dim+i] - x_in[i], 2.0 );
	if( sum > obj_in->dlt[idx_in] * obj_in->dlt[idx_in] )
		return 0.0;

	/* NOTE: Change this to punity_neighbors_fast() */
	sum = 0.0;
	for(i=0;i<obj_in->npts;i++)
	{
		res = 0.0;
		for(j=0;j<obj_in->dim;j++)
			vec[j] = x_in[j] - obj_in->pts[i*obj_in->dim+j], res += vec[j] * vec[j];
		res = sqrt( res );
		if( res < obj_in->dlt[i] )
		{
			if( obj_in->bdry[i] == 0 )
				sum += obj_in->wfs( obj_in->dim, obj_in->dlt[i], vec );
			else
				sum += obj_in->wfs( obj_in->dim, obj_in->dlt[i], vec ) / pow( res / obj_in->dlt[i], (double) exp_in );
		}
	}
	res = 0.0;
	for(i=0;i<obj_in->dim;i++)
		vec[i] = x_in[i] - obj_in->pts[idx_in*obj_in->dim+i], res += vec[i] * vec[i];
	res = sqrt( res );
	if( res > obj_in->dlt[idx_in] )
		return 0.0;
	else
	{
		if( obj_in->bdry[idx_in] == 0 )
			y = obj_in->wfs( obj_in->dim, obj_in->dlt[idx_in], vec ) / sum;
		else
			y = obj_in->wfs( obj_in->dim, obj_in->dlt[idx_in], vec ) / pow( res / obj_in->dlt[idx_in], (double) exp_in ) / sum;
	}
	free( vec );
	return y;
}

/**
 * Evaluate the derivative of r = | xi - xj | w.r.t. drv_in; FIXME: This function
 * returns values which become singular as r -> 0; figure out a way to get around this
 * @param dim_in Dimension of the vector space
 * @param drv_in Derivative integer vector
 * @param x_in Point at which to evaluate the derivative
 */
__device__ double radial_deriv_evaluate( int dim_in, int *drv_in, double *x_in )
{
	int b,c,i,j,p,n,*v,*w,*s,*m;
	double d,prd,sum = 0.0;

	/* Set up the vector to partition */
	n = 0;
	for(i=0;i<dim_in;i++)
		n += drv_in[i];
	v = (int*) malloc( n * sizeof(int) );
	w = (int*) malloc( n * sizeof(int) );
	s = (int*) malloc( n * sizeof(int) );
	m = (int*) malloc( n * sizeof(int) );
	for(i=0,p=0;i<dim_in;i++)
		for(j=0;j<drv_in[i];j++)
			v[p++] = i;

	/* Start iterating through the partitions */
	partition_init( s, m, n );
	do
	{
		/* Variable b contains the number of blocks in this partition */
		for(i=0;i<n;i++)
			if( i == 0 || s[i] > b )
				b = s[i];

		/* Generate each of the blocks from the membership vector s */
		prd = 1.0;
		for(i=0;i<b;i++) /* Index i is the current block */
		{
			/* The number of entries in block i is counted in c */
			for(j=0,c=0;j<n;j++)
				if( s[j] == i + 1 )
					w[c++] = v[j]; /* Take derivative of u = x1^2 + ... + xd^2 w.r.t. coordinate v[j] */

			/* Do the derivative */
			if( c > 2 )
				d = 0.0;
			else
			{
				if( c == 2 )
				{
					if( w[0] != w[1] )
						d = 0.0;
					else
						d = 2.0;
				}
				else if( c == 1 )
					d = 2.0 * x_in[w[0]];
				else /* The zero derivative should not occur in this sequence, but anyway... */
				{
					d = 0.0;
					for(j=0;j<dim_in;j++)
						d += x_in[j] * x_in[j];
				}
			}

			/* Multiply the derivative in d for this block into the total product */
			prd *= d; /* Variable d is derivative of u consistent with this block */
		}

		/* Calculate b derivative of radial function */
		for(i=0;i<b;i++)
			prd *= ( 0.5 - (double) i );
		d = 0.0;
		for(i=0;i<dim_in;i++)
			d += x_in[i] * x_in[i];
		prd *= pow( d, 0.5 - (double) b );
		sum += prd;
	}
	while( partition_next( s, m, n ) != 0 );

	free( v );
	free( w );
	free( s );
	free( m );

	return sum;
}

/**
 * Evaluates the derivatives of r**-p to any order in any dimension
 * using radial_deriv_evaluate
 * @param dim_in Dimension of vector space
 * @param drv_in The derivative integer vector
 * @param x_in Point at which to evaluate derivative
 * @param a_in Dilation factor to use; r -> (r/a)**-p
 * @param exp_in Exponent p to use
 */
__device__ double rational_radial_deriv_evaluate( int dim_in, int *drv_in, double *x_in, double a_in, int exp_in )
{
	int b,c,i,j,k,p,n,*v,*w,*s,*m,*q;
	double d,y,prd,sum = 0.0;

	/* Set up the vector to partition */
	n = 0;
	for(i=0;i<dim_in;i++)
		n += drv_in[i];
	v = (int*) malloc( n * sizeof(int) );
	w = (int*) malloc( n * sizeof(int) );
	s = (int*) malloc( n * sizeof(int) );
	m = (int*) malloc( n * sizeof(int) );
	q = (int*) malloc( dim_in * sizeof(int) );
	for(i=0,p=0;i<dim_in;i++)
		for(j=0;j<drv_in[i];j++)
			v[p++] = i;

	/* Form radial value */
	d = 0.0;
	for(i=0;i<dim_in;i++)
		d += x_in[i] * x_in[i];
	d = sqrt( d );

	/* Return now if derivative order is zero */
	if( n == 0 )
	{
		free( v );
		free( w );
		free( s );
		free( m );
		free( q );
		return pow( d / a_in, (double) ( -exp_in ) );
	}

	/* Start iterating through the partitions */
	partition_init( s, m, n );
	do
	{
		/* Variable b contains the number of blocks in this partition */
		for(i=0;i<n;i++)
			if( i == 0 || s[i] > b )
				b = s[i];

		/* Take the b-th derivative of r**-p w.r.t. r because b is the number of blocks in partition s */
		prd = 1.0;
		for(i=0;i<b;i++)
			prd *= (double) ( -exp_in - i );
		prd *= pow( d, (double) ( -exp_in - b ) );

		/* Now deal with all the other block derivatives of r w.r.t. x's */
		for(i=0;i<b;i++)
		{
			/* The number of entries in block i is counted in c */
			for(j=0,c=0;j<n;j++)
				if( s[j] == i + 1 )
					w[c++] = v[j]; /* Take derivative of u = x1^2 + ... + xd^2 w.r.t. coordinate v[j] */

			/* Now rebuild the derivative in drv_in format to pass to radial_deriv_evaluate */
			for(j=0;j<dim_in;j++)
				q[j] = 0;
			for(j=0;j<c;j++)
				q[w[j]]++; /* Everytime an index appears in w, increment its component once */
			y = radial_deriv_evaluate( dim_in, q, x_in );
			prd *= y;
		}
		sum += prd;
	}
	while( partition_next( s, m, n ) != 0 );

	free( v );
	free( w );
	free( s );
	free( m );
	free( q );

	return pow( a_in, (double) exp_in ) * sum;
}

/**
 * Evaluate the derivative of phi_i / ( sum_j phi_j ) w.r.t. the phi_j's themselves
 * where each phi_j is evaluated at the input point x_in
 */
__device__ double punity_comp_deriv_evaluate( punity_t *obj_in, int idx_in, int *drv_in, int q_in, int *nb_in, double *x_in )
{
	int i,j,k,n;
	double sum,prd1,prd2;

	/* Build the rank of the derivative w.r.t. all variables not equal to idx_in */
	n = 0;
	for(i=0;i<q_in;i++)
		n += drv_in[i];

	/* Which position in drv_in corresponds to idx_in */
	for(i=0,k=-1;i<q_in;i++)
		if( nb_in[i] == idx_in )
			k = i;
	if( k == -1 ) /* Zero/error because idx_in is not a neighbor to x_in */
		return 0.0;

	/* Calculate sum of all window functions */
	sum = 0.0;
        for(i=0;i<q_in;i++)
                sum += punity_window_evaluate( obj_in, nb_in[i], x_in );

	/* Simplified derivative only has two terms! */
	prd1 = punity_window_evaluate( obj_in, idx_in, x_in ) * ( n % 2 == 0 ? 1.0 : -1.0 )
		* (double) factorial( n ) * pow( sum, -1.0 * (double) ( n + 1 ) );

	/* If drv_in[k] == 0 then return prd1 as output */
	if( drv_in[k] == 0 )
		return prd1;

	/* Build the second term in the sum */
	prd2 = (double) drv_in[k] * ( ( n - 1 ) % 2 == 0 ? 1.0 : -1.0 ) * (double) factorial( n - 1 ) * pow( sum, -1.0 * (double) n );

	/* Form sum */
	return prd1 + prd2;
}

/**
 * This functions makes use of the Faa di Bruno formula for higher derivatives
 * of a composition of an r-dependent function with r as a function of
 * the individual x variables; r = sqrt( x1^2 + ... + xd^2 )
 */
__device__ double punity_window_deriv_evaluate( punity_t *obj_in, int idx_in, int *drv_in, double *x_in )
{
	int b,c,i,j,p,n,*v,*w,*s,*m,*t;
        double d,prd,sum = 0.0;

	/* Set up the vector to partition */
        n = 0;
        for(i=0;i<obj_in->dim;i++)
                n += drv_in[i];
        v = (int*) malloc( n * sizeof(int) );
        w = (int*) malloc( n * sizeof(int) );
        s = (int*) malloc( n * sizeof(int) );
        m = (int*) malloc( n * sizeof(int) );
	t = (int*) malloc( obj_in->dim * sizeof(int) );
        for(i=0,p=0;i<obj_in->dim;i++)
                for(j=0;j<drv_in[i];j++)
                        v[p++] = i;

	/* If derivative order is zero then return the function undifferentiated */
	if( n == 0 )
	{
		free( v );
		free( w );
		free( s );
		free( m );
		free( t );
		return obj_in->wfs( obj_in->dim, obj_in->dlt[idx_in], x_in );
	}

	/* Start iterating through the partitions */
        partition_init( s, m, n );
        do
        {
                /* Variable b contains the number of blocks in this partition */
                for(i=0;i<n;i++)
                        if( i == 0 || s[i] > b )
                                b = s[i];

                /* Generate each of the blocks from the membership vector s */
                prd = 1.0;
                for(i=0;i<b;i++) /* Index i is the current block */
                {
                        /* The number of entries in block i is counted in c */
                        for(j=0,c=0;j<n;j++)
                                if( s[j] == i + 1 )
                                        w[c++] = v[j]; /* Take derivative of u = x1^2 + ... + xd^2 w.r.t. coordinate v[j] */

			/* Build the derivative in terms of exponents */
			for(j=0;j<obj_in->dim;j++)
				t[j] = 0;
			for(j=0;j<c;j++)
				t[w[j]]++; /* Everytime an index appears in w, increment its component once */

                        /* Do the derivative */
			d = radial_deriv_evaluate( obj_in->dim, t, x_in ); /* No dilation factor here; all contained in wfsd */

                        /* Multiply the derivative in d for this block into the total product */
                        prd *= d; /* Variable d is derivative of u consistent with this block */
                }

		/* Calculate the b order derivative of the radial polynomial */
		prd *= obj_in->wfsd( obj_in->dim, b, obj_in->dlt[idx_in], x_in );

		/* Add the contribution from this partition to the total in sum */
                sum += prd;
	}
	while( partition_next( s, m, n ) != 0 );

	free( v );
	free( w );
	free( s );
	free( m );
	free( t );

	return sum;
}

/**
 * Evaluate arbitrary derivatives of partition of unity functions with
 * a singularity of order exp_in at the origin of the function
 */
__device__ double punity_window_deriv_evaluate_delta( punity_t *obj_in, int idx_in, int *drv_in, double *x_in, int exp_in )
{
	int i,j,k,m,n,p,q,r,*qc,*qd,*v,*cmb,*dmb;
	double d,prd,sum = 0.0;

	/* Do some setup */
	n = 0;
	for(i=0;i<obj_in->dim;i++)
		n += drv_in[i];
	v = (int*) malloc( n * sizeof(int) );
	qc = (int*) malloc( obj_in->dim * sizeof(int) );
	qd = (int*) malloc( obj_in->dim * sizeof(int) ); /* Complement of q w.r.t. the current cmb state */
	cmb = (int*) malloc( n * sizeof(int) );
	dmb = (int*) malloc( n * sizeof(int) );

	/* Calculate the derivative vector; a sequence of the numbers 0,...,dim-1 */
	for(i=0,p=0;i<obj_in->dim;i++)
		for(j=0;j<drv_in[i];j++)
			v[p++] = i;

	/* Calculate the radial vector */
	d = 0.0;
	for(i=0;i<obj_in->dim;i++)
		d += x_in[i] * x_in[i];
	d = sqrt( d );

	/* If derivative order is zero then return the function undifferentiated */
	if( n == 0 )
	{
		free( v );
		free( qc );
		free( qd );
		free( cmb );
		free( dmb );
		return obj_in->wfs( obj_in->dim, obj_in->dlt[idx_in], x_in ) / pow( d / obj_in->dlt[idx_in], (double) exp_in );
	}

	/* Iterate through all combinations sizes and for each size, iterate through all combinations */
	for(i=0;i<n;i++)
	{
		combinadic_init( n, i, cmb );
		k = binomial( n, i );
		for(j=0;j<k;j++)
		{
			/* Form the derivative vector for this case */
			for(m=0;m<obj_in->dim;m++)
				qc[m] = 0;
			for(m=0;m<i;m++)
				qc[v[cmb[m]]]++;

			/* Build the complement */
			if( i > 0 )
			{
				for(m=0;m<cmb[0];m++)
					dmb[m] = m;
				for(m=0,r=cmb[0];m<i-1;m++) /* Iterate consecutive integer pairs */
					for(q=cmb[m]+1;q<cmb[m+1];q++)
						dmb[r++] = q;
				for(m=cmb[i-1]+1;m<n;m++)
					dmb[r++] = m;
			}
			else
				for(m=0;m<n;m++)
					dmb[m] = m;

			/* Form the derivative of the window */
			for(m=0;m<obj_in->dim;m++)
				qd[m] = 0;
			for(m=0;m<n-i;m++)
				qd[v[dmb[m]]]++;

			/* Apply the derivative to the window function and evaluate it */
			prd = 1.0;
			if( i == 0 )
				prd *= pow( d, (double) ( -exp_in ) );
			else
				prd *= rational_radial_deriv_evaluate( obj_in->dim, qc, x_in, obj_in->dlt[idx_in], exp_in );
			if( n - i == 0 )
				prd *= obj_in->wfs( obj_in->dim, obj_in->dlt[idx_in], x_in );
			else
				prd *= punity_window_deriv_evaluate( obj_in, idx_in, qd, x_in );

			/* Add this shit up */
			sum += prd;

			/* Take a step to the next combinadic vector */
			combinadic_next( n, i, cmb );
		}
	}

	free( v );
	free( qc );
	free( qd );
	free( cmb );
	free( dmb );

	return sum;
}

__device__ int punity_neighbors( punity_t *obj_in, double *x_in, int **nb_out )
{
	int i,j,q,na,*nb,*nc;
	double prd;

	na = PUNITY_NB_INCREMENT;
        nb = (int*) malloc( na * sizeof(int) );
        for(i=0,q=0;i<obj_in->npts;i++)
        { 
                prd = 0.0;
                for(j=0;j<obj_in->dim;j++)
                        prd += pow( obj_in->pts[i*obj_in->dim+j] - x_in[j], 2.0 );
                prd = sqrt( prd );
                if( prd < obj_in->dlt[i] ) /* Then it is close enough to contribute */
                {
                        /* See if we need to allocate more space */
                        if( q + 1 > na )
                        {
                                na += PUNITY_NB_INCREMENT;
				nc = (int*) malloc( na * sizeof(int) );
				for(j=0;j<q;j++)
					nc[j] = nb[j];
				free( nb );
				nb = nc; /* Swap definitions to simulate realloc */
                        }
                        nb[q++] = i;
                }
        }
	*nb_out = nb;

	return q;
}

/**
 * Evaluate the derivative of a partition-of-unity function with given
 * index and given components
 * @param obj_in Partition of unity object
 * @param idx_in Index of the PU functions to evaluate (index to the point)
 * @param drv_in Derivative specified as the order w.r.t. each independent variable
 * @param x_in Point at which to evaluate the resulting derivative
 * @param nb_in Neighbors of point idx_in; if NULL it will be calculated
 * @return The value of the derivative at point x_in
 */
__device__ double punity_term_deriv_evaluate( punity_t *obj_in, int idx_in, int *drv_in, double *x_in, int *nb_in, int q_in )
{
	int b,c,i,j,k,p,q,n,bf,*v,*w,*s,*m,*t,*u,*nb;
	long *size,*index;
        double d,prd,tsum,*y,sum = 0.0;

	/* First check to see if x_in is too far from obj_in->pts + idx_in * dim */
	prd = 0.0;
	for(i=0;i<obj_in->dim;i++)
		prd += pow( obj_in->pts[idx_in*obj_in->dim+i] - x_in[i], 2.0 );
	prd = sqrt( prd );
	if( prd > obj_in->dlt[idx_in] )
		return 0.0;

	/* Temporary position vector for translation */
	y = (double*) malloc( obj_in->dim * sizeof(double) );

	/* Set up the vector to partition */
        n = 0;
        for(i=0;i<obj_in->dim;i++)
                n += drv_in[i];
        v = (int*) malloc( n * sizeof(int) );
        w = (int*) malloc( n * sizeof(int) );
        s = (int*) malloc( n * sizeof(int) );
        m = (int*) malloc( n * sizeof(int) );
	t = (int*) malloc( obj_in->dim * sizeof(int) );
        for(i=0,p=0;i<obj_in->dim;i++)
                for(j=0;j<drv_in[i];j++)
                        v[p++] = i;

	/* Let q count the number of functions contributing to the total particle function idx_in */
#ifdef PUNITY_USE_KDTREES
	if( nb_in == NULL )
		q = punity_neighbors_fast( obj_in, x_in, &nb ), bf = 1; /* Free nb at the end */
	else
		q = q_in, nb = nb_in, bf = 0; /* Don't free at the end */
#else
	if( nb_in == NULL )
		q = punity_neighbors( obj_in, x_in, &nb ), bf = 1; /* Free nb at the end */
	else
		q = q_in, nb = nb_in, bf = 0; /* Don't free nb at the end */
#endif
	u = (int*) malloc( q * sizeof(int) ); /* Serves as derivative vector */

	/* Maximum number of blocks possible is n */
	size = (long*) malloc( n * sizeof(long) );
	index = (long*) malloc( n * sizeof(long) );
	for(i=0;i<n;i++)
		size[i] = q - 1; /* Maximum value is q - 1; starting at 0 makes for q values */

	/* Start iterating through the partitions */
        partition_init( s, m, n );
        do
        {
                /* Variable b contains the number of blocks in this partition, s */
                for(i=0;i<n;i++)
                        if( i == 0 || s[i] > b )
                                b = s[i];

		/* Iterate through all length-b vectors with entries in {1,...,q} */
		for(i=0;i<b;i++)
			index[i] = 0;
		do
		{
			/* Build the derivative integer vector from the index vector in this loop */
			for(i=0;i<q;i++)
				u[i] = 0;
			for(i=0;i<b;i++)
				u[index[i]]++;

			/* Calculate the derivative w.r.t. the window functions */
			prd = punity_comp_deriv_evaluate( obj_in, idx_in, u, q, nb, x_in );

	                /* Generate each of the blocks from the membership vector */
	                for(i=0;i<b;i++) /* Index i is the current block */
	                {
	                        /* The number of entries in block i is counted in c */
	                        for(j=0,c=0;j<n;j++)
	                                if( s[j] == i + 1 )
	                                        w[c++] = v[j]; /* Take derivative of function w.r.t. coordinate v[j] */
				for(j=0;j<obj_in->dim;j++)
					t[j] = 0;
				for(j=0;j<c;j++)
					t[w[j]]++;

				/* Build the product of partition derivatives specified by the indexes in index of each block */
				for(j=0;j<obj_in->dim;j++)
					y[j] = x_in[j] - obj_in->pts[nb[index[i]]*obj_in->dim+j];
				tsum = punity_window_deriv_evaluate( obj_in, nb[index[i]], t, y );
				prd *= tsum;
	                }
			sum += prd;
		}
		while( arraynext( b, size, index ) != -1 );
	}
	while( partition_next( s, m, n ) != 0 );

	free( v );
	free( w );
	free( s );
	free( m );
	free( t );
	free( u );
	free( y );
	free( size ); free( index );
	if( bf )
		free( nb );

	return sum;
}

/**
 * Evaluate the derivative of a partition-of-unity function with given
 * index and given components
 * @param obj_in Partition of unity object
 * @param idx_in Index of the PU functions to evaluate (index to the point)
 * @param drv_in Derivative specified as the order w.r.t. each independent variable
 * @param x_in Point at which to evaluate the resulting derivative
 * @param exp_in Exponent to use for singularity, if it exists
 * @param nb_in Neighbors given as input; NULL means calculate it yourself
 * @return The value of the derivative at point x_in
 */
__device__ double punity_term_deriv_evaluate_delta( punity_t *obj_in, int idx_in, int *drv_in, double *x_in, int exp_in, int *nb_in, int q_in )
{
	int b,c,i,j,k,p,q,n,bf,*v,*w,*s,*m,*t,*u,*nb;
	long *size,*index;
        double d,prd,tsum,*y,sum = 0.0;

	/* First check to see if x_in is too far from obj_in->pts + idx_in * dim */
	prd = 0.0;
	for(i=0;i<obj_in->dim;i++)
		prd += pow( obj_in->pts[idx_in*obj_in->dim+i] - x_in[i], 2.0 );
	prd = sqrt( prd );
	if( prd > obj_in->dlt[idx_in] )
		return 0.0;

	/* Temporary position vector for translation */
	y = (double*) malloc( obj_in->dim * sizeof(double) );

	/* Set up the vector to partition */
        n = 0;
        for(i=0;i<obj_in->dim;i++)
                n += drv_in[i];
        v = (int*) malloc( n * sizeof(int) );
        w = (int*) malloc( n * sizeof(int) );
        s = (int*) malloc( n * sizeof(int) );
        m = (int*) malloc( n * sizeof(int) );
	t = (int*) malloc( obj_in->dim * sizeof(int) );
        for(i=0,p=0;i<obj_in->dim;i++)
                for(j=0;j<drv_in[i];j++)
                        v[p++] = i;

	/* Let q count the number of functions contributing to the total particle function idx_in */
#ifdef PUNITY_USE_KDTREES
	if( nb_in == NULL )
		q = punity_neighbors_fast( obj_in, x_in, &nb ), bf = 1;
	else
		q = q_in, nb = nb_in, bf = 0;
#else
	if( nb_in == NULL )
		q = punity_neighbors( obj_in, x_in, &nb ), bf = 1;
	else
		q = q_in, nb = nb_in, bf = 0;
#endif

	u = (int*) malloc( q * sizeof(int) ); /* Serves as derivative vector */

	/* Maximum number of blocks possible is n */
	size = (long*) malloc( n * sizeof(long) );
	index = (long*) malloc( n * sizeof(long) );
	for(i=0;i<n;i++)
		size[i] = q - 1; /* Maximum value is q - 1; starting at 0 makes for q values */

	/* Start iterating through the partitions */
        partition_init( s, m, n );
        do
        {
                /* Variable b contains the number of blocks in this partition, s */
                for(i=0;i<n;i++)
                        if( i == 0 || s[i] > b )
                                b = s[i];

		/* Iterate through all length-b vectors with entries in {1,...,q} */
		for(i=0;i<b;i++)
			index[i] = 0;
		do
		{
			/* Build the derivative integer vector from the index vector in this loop */
			for(i=0;i<q;i++)
				u[i] = 0;
			for(i=0;i<b;i++)
				u[index[i]]++;

			/* Calculate the derivative w.r.t. the window functions */
			prd = punity_comp_deriv_evaluate( obj_in, idx_in, u, q, nb, x_in );

	                /* Generate each of the blocks from the membership vector */
	                for(i=0;i<b;i++) /* Index i is the current block */
	                {
	                        /* The number of entries in block i is counted in c */
	                        for(j=0,c=0;j<n;j++)
	                                if( s[j] == i + 1 )
	                                        w[c++] = v[j]; /* Take derivative of function w.r.t. coordinate v[j] */
				for(j=0;j<obj_in->dim;j++)
					t[j] = 0;
				for(j=0;j<c;j++)
					t[w[j]]++;

				/* Build the product of partition derivatives specified by the indexes in index of each block */
				for(j=0;j<obj_in->dim;j++)
					y[j] = x_in[j] - obj_in->pts[nb[index[i]]*obj_in->dim+j];
				if( obj_in->bdry[nb[index[i]]] == 0 )
					tsum = punity_window_deriv_evaluate( obj_in, nb[index[i]], t, y );
				else
					tsum = punity_window_deriv_evaluate_delta( obj_in, nb[index[i]], t, y, exp_in );
				prd *= tsum;
	                }
			sum += prd;
		}
		while( arraynext( b, size, index ) != -1 );
	}
	while( partition_next( s, m, n ) != 0 );

	free( v );
	free( w );
	free( s );
	free( m );
	free( t );
	free( u );
	free( y );
	free( size ); free( index );
	if( bf )
		free( nb );

	return sum;
}

/**
 * This function is intended to be a faster version of punity_term_deriv_evaluate
 * for use calculating the first derivatives only
 */
__device__ double punity_term_first_deriv_evaluate( punity_t *obj_in, int idx_in, int drv_in, double *x_in )
{
	
}

/**
 * Evaluate particle-localized polynomial term
 */
__device__ double punity_eval_local_poly( punity_t *obj_in, int idx_in, int *pidx_in, double *x_in )
{
	int i;
	double prd = 1.0;

	/* Calculate the monomial term evaluated at x_in */
	for(i=0;i<obj_in->dim;i++) /* Evaluate everything on the local scale as well */
		prd *= pow( ( x_in[i] - obj_in->pts[idx_in*obj_in->dim+i] ) / obj_in->dlt[idx_in], (double) pidx_in[i] );

	return prd * punity_evaluate( obj_in, idx_in, x_in );
}

/**
 * Evaluate particle-localizaed polynomial term while observing
 * singular particle terms
 */
__device__ double punity_eval_local_poly_delta( punity_t *obj_in, int idx_in, int *pidx_in, double *x_in, int exp_in )
{
	int i;
	double prd = 1.0;

	/* Calculate the monomial term evaluated at x_in */
	for(i=0;i<obj_in->dim;i++)
		prd *= pow( ( x_in[i] - obj_in->pts[idx_in*obj_in->dim+i] ) / obj_in->dlt[idx_in], (double) pidx_in[i] );

	return prd * punity_evaluate_delta( obj_in, idx_in, x_in, exp_in );
}

/**
 * Evaluate derivatives of the particle-localized polynomial term
 * while ignoring singular particles
 */
__device__ double punity_term_deriv_local_poly( punity_t * obj_in, int idx_in, int *pidx_in, int *drv_in, double *x_in, int *nb_in, int q_in )
{
	int i,j,k,m,n,p,q,r,*qc,*qd,*v,*cmb,*dmb;
	double prd,coeff,sum = 0.0;

	/* Do some setup */
	n = 0;
	for(i=0;i<obj_in->dim;i++)
		n += drv_in[i];
	v = (int*) malloc( n * sizeof(int) );
	qc = (int*) malloc( obj_in->dim * sizeof(int) );
	qd = (int*) malloc( obj_in->dim * sizeof(int) ); /* Complement of q w.r.t. the current cmb state */
	cmb = (int*) malloc( n * sizeof(int) );
	dmb = (int*) malloc( n * sizeof(int) );

	/* Calculate the derivative vector; a sequence of the numbers 0,...,dim-1 */
	for(i=0,p=0;i<obj_in->dim;i++)
		for(j=0;j<drv_in[i];j++)
			v[p++] = i;

	/* If derivative order is zero then return the function undifferentiated */
	if( n == 0 )
	{
		free( v );
		free( qc );
		free( qd );
		free( cmb );
		free( dmb );
		return punity_eval_local_poly( obj_in, idx_in, pidx_in, x_in );
	}

	/* Iterate through all combinations sizes and for each size, iterate through all combinations */
	for(i=0;i<n;i++)
	{
		combinadic_init( n, i, cmb );
		k = binomial( n, i );
		for(j=0;j<k;j++)
		{
			/* Form the derivative vector for this case */
			for(m=0;m<obj_in->dim;m++)
				qc[m] = 0;
			for(m=0;m<i;m++)
				qc[v[cmb[m]]]++;

			/* Build the complement */
			if( i > 0 )
			{
				for(m=0;m<cmb[0];m++)
					dmb[m] = m;
				for(m=0,r=cmb[0];m<i-1;m++) /* Iterate consecutive integer pairs */
					for(q=cmb[m]+1;q<cmb[m+1];q++)
						dmb[r++] = q;
				for(m=cmb[i-1]+1;m<n;m++)
					dmb[r++] = m;
			}
			else
				for(m=0;m<n;m++)
					dmb[m] = m;

			/* Form the derivative of the window */
			for(m=0;m<obj_in->dim;m++)
				qd[m] = 0;
			for(m=0;m<n-i;m++)
				qd[v[dmb[m]]]++;

			/* Apply the derivative to the window function and evaluate it */
			prd = 1.0;
			if( i == 0 )
			{
				/* Put the value of the polynomial term in pidx_in in prd */
				for(m=0;m<obj_in->dim;m++)
					prd *= pow( ( x_in[m] - obj_in->pts[idx_in*obj_in->dim+m] ) / obj_in->dlt[idx_in], (double) pidx_in[m] );
			}
			else
			{
				/* Go through and do the derivative of the polynomial */
				for(m=0;m<obj_in->dim;m++)
				{
					p = pidx_in[m] - qc[m]; /* If qc[j] > pidx_in[j] then the whole thing is zero */
					if( p < 0 )
					{
						prd = 0.0;
						break; /* Break here because if one partial is zero then the whole term is; no need to do it */
					}
					else
					{
						prd *= pow( ( x_in[m] - obj_in->pts[idx_in*obj_in->dim+m] ) / obj_in->dlt[idx_in], (double) p ) / pow( obj_in->dlt[idx_in], (double) i );
						for(p=0;p<qc[m];p++)
							prd *= (double) ( pidx_in[m] - p );
					}
				}
			}
			if( n - i == 0 )
				prd *= punity_evaluate( obj_in, idx_in, x_in );
			else /* Evaluate the derivative of the partition of unity function */
			{
				prd *= punity_term_deriv_evaluate( obj_in, idx_in, qd, x_in, nb_in, q_in );
			}

			/* Add this shit up */
			sum += prd;

			/* Take a step to the next combinadic vector */
			combinadic_next( n, i, cmb );
		}
	}

	free( v );
	free( qc );
	free( qd );
	free( cmb );
	free( dmb );

	return sum;
}

/**
 * Evaluate derivatives of the particle-localized polynomial term
 * while ignoring singular particles
 */
__device__ double punity_term_deriv_local_poly_delta( punity_t * obj_in, int idx_in, int *pidx_in, int *drv_in, double *x_in, int exp_in, int *nb_in, int q_in )
{
	int i,j,k,m,n,p,q,r,*qc,*qd,*v,*cmb,*dmb;
	double prd,coeff,sum = 0.0;

	/* Do some setup */
	n = 0;
	for(i=0;i<obj_in->dim;i++)
		n += drv_in[i];
	v = (int*) malloc( n * sizeof(int) );
	qc = (int*) malloc( obj_in->dim * sizeof(int) );
	qd = (int*) malloc( obj_in->dim * sizeof(int) ); /* Complement of q w.r.t. the current cmb state */
	cmb = (int*) malloc( n * sizeof(int) );
	dmb = (int*) malloc( n * sizeof(int) );

	/* Calculate the derivative vector; a sequence of the numbers 0,...,dim-1 */
	for(i=0,p=0;i<obj_in->dim;i++)
		for(j=0;j<drv_in[i];j++)
			v[p++] = i;

	/* If derivative order is zero then return the function undifferentiated */
	if( n == 0 )
	{
		free( v );
		free( qc );
		free( qd );
		free( cmb );
		free( dmb );
		return punity_eval_local_poly_delta( obj_in, idx_in, pidx_in, x_in, exp_in );
	}

	/* Iterate through all combinations sizes and for each size, iterate through all combinations */
	for(i=0;i<n;i++)
	{
		combinadic_init( n, i, cmb );
		k = binomial( n, i );
		for(j=0;j<k;j++)
		{
			/* Form the derivative vector for this case */
			for(m=0;m<obj_in->dim;m++)
				qc[m] = 0;
			for(m=0;m<i;m++)
				qc[v[cmb[m]]]++;

			/* Build the complement */
			if( i > 0 )
			{
				for(m=0;m<cmb[0];m++)
					dmb[m] = m;
				for(m=0,r=cmb[0];m<i-1;m++) /* Iterate consecutive integer pairs */
					for(q=cmb[m]+1;q<cmb[m+1];q++)
						dmb[r++] = q;
				for(m=cmb[i-1]+1;m<n;m++)
					dmb[r++] = m;
			}
			else
				for(m=0;m<n;m++)
					dmb[m] = m;

			/* Form the derivative of the window */
			for(m=0;m<obj_in->dim;m++)
				qd[m] = 0;
			for(m=0;m<n-i;m++)
				qd[v[dmb[m]]]++;

			/* Apply the derivative to the window function and evaluate it */
			prd = 1.0;
			if( i == 0 )
			{
				/* Put the value of the polynomial term in pidx_in in prd */
				for(m=0;m<obj_in->dim;m++)
					prd *= pow( ( x_in[m] - obj_in->pts[idx_in*obj_in->dim+m] ) / obj_in->dlt[idx_in], (double) pidx_in[m] );

			}
			else
			{
				/* Go through and do the derivative of the polynomial */
				for(m=0;m<obj_in->dim;m++)
				{
					p = pidx_in[m] - qc[m]; /* If qc[j] > pidx_in[j] then the whole thing is zero */
					if( p < 0 )
					{
						prd = 0.0;
						break; /* Break here because if one partial is zero then the whole term is; no need to do it */
					}
					else
					{
						prd *= pow( ( x_in[m] - obj_in->pts[idx_in*obj_in->dim+m] ) / obj_in->dlt[idx_in], (double) p ) / pow( obj_in->dlt[idx_in], (double) i );
						for(p=0;p<qc[m];p++)
							prd *= (double) ( pidx_in[m] - p );
					}
				}
			}
			if( n - i == 0 )
				prd *= punity_evaluate_delta( obj_in, idx_in, x_in, exp_in );
			else /* Evaluate the derivative of the partition of unity function */
				prd *= punity_term_deriv_evaluate_delta( obj_in, idx_in, qd, x_in, exp_in, nb_in, q_in );

			/* Add this shit up */
			sum += prd;

			/* Take a step to the next combinadic vector */
			combinadic_next( n, i, cmb );
		}
	}

	free( v );
	free( qc );
	free( qd );
	free( cmb );
	free( dmb );

	return sum;
}

/**
 * Kernel function to generate the overlap and Hamiltonian matrices
 */
__global__ void puksham_mbuild_cuda( punity_t *pu, int mdim, int nbp, shape_t *dm, int *nlbase, int **lbase, int *ltg, nuclei_t *nuc, int quadn, double *qpts, double *qwts,
					 long **jap, double **Ap, long **jbp, double **Bp, long cc, long c_spmat_inc,
						int b_use_external_potential, int b_load_overl_mat, int b_load_stiff_mat,
							int b_use_singular, int i_sing_order, int *b_have_stiff_mat, int *b_have_overl_mat )
{
	/* Counting variables */
	int i,ii,j,jj,k,m,p,pp,q,qq,ret;

	/* Important stuff */
	int dim = pu->dim;
	int npts = pu->npts;
	int *nb,nnb;
	int *expl,*expr,*drv;
	int idx;
	long *ia = *iap, *ja = *jap;
	long *ib = *ibp, *jb = *jbp;
	long *size,*index;
	double *A = *Ap, *B = *Bp;
	double *ctr;
	double x1,x2,y,rad;
	double *qbox,*nqbox;
	double qw,*qx;
	double rum,sum,tum;

	/* Allocate stuff */
	qbox = (double*) malloc( dim * dim * sizeof(double) );
	nqbox = (double*) malloc( dim * dim * sizeof(double) );
	ctr = (double*) malloc( dim * sizeof(double) );
	qx = (double*) malloc( dim * sizeof(double) );
	size = (long*) malloc( dim * sizeof(long) );
	index = (long*) malloc( dim * sizeof(long) );
	expl = (int*) malloc( dim * sizeof(int) );
	expr = (int*) malloc( dim * sizeof(int) );
	drv = (int*) malloc( dim * sizeof(int) );

	/* Initialize the matrices */
	for(i=0;i<mdim;i++)
		ia[i] = -1, ib[i] = -1;
	p = 0, pp = 0;
	for(i=0;i<npts-nbp;i++)
	{
		if( npts >= 20 )
		{
			for(j=0;j<i/(npts/20);j++)
				fprintf( stderr, "=" );
			fprintf( stderr, ">%3d%%", i * 100 / ( npts - nbp ) );
		}
		for(ii=0;ii<nlbase[ltg[i]];ii++)
		{
			/* Mark the start of the diagonal entry in the global scheme */
			qq = pp;

			/* Iterate through second function to calculate inner product */
			for(j=i;j<npts-nbp;j++)
			{
				for(jj=(j==i?ii:0);jj<nlbase[ltg[j]];jj++)
				{
					if( sphere_intersection( dim, pu->pts + ltg[i] * dim, pu->dlt[ltg[i]], pu->pts + ltg[j] * dim, pu->dlt[ltg[j]], ctr, &rad, qbox ) == 1 )
					{
						/* Build the local basis for the intersection of spheres */
						for(k=0;k<dim;k++)
						{
							sum = 0.0;
							for(m=0;m<dim;m++)
								sum += qbox[k*dim+m] * qbox[k*dim+m];
							sum = sqrt( sum );
							for(m=0;m<dim;m++)
								nqbox[k*dim+m] = qbox[k*dim+m] / sum;
						}

						/* Calculate the entries of the inner product matrix */
						for(k=0;k<dim;k++)
							index[k] = 0, size[k] = quadn - 1;
						rum = 0.0, sum = 0.0, tum = 0.0;
						do
						{
							lens_gauss_point( dim, pu->pts + ltg[i] * dim, pu->dlt[ltg[i]], pu->pts + ltg[j] * dim, pu->dlt[ltg[j]],
								rad, nqbox, index, qpts, qwts, qx, &qw );
							domain_indicator( qx, dm, &ret );
							if( ret == 1 )
							{
								/* Generate the neighbors list for point qx */
								nnb = punity_neighbors( pu, qx, &nb );

								/* Use nb to form these values */
								if( !b_load_overl_mat || !b_load_stiff_mat )
								{
									if( lbase[ltg[i]][ii] >= 0 )
									{
										global_polynomial_vector( lbase[ltg[i]][ii], dim, expl );
										if( b_use_singular )
											x1 = punity_eval_local_poly_delta( pu, ltg[i], expl, qx, i_sing_order );
										else
											x1 = punity_eval_local_poly( pu, ltg[i], expl, qx );
									}
									else
									{
										/* Deal with case in which lbase[ltg[i]][ii] < 0 */
										idx = 1 - idx;
										
									}
									if( lbase[ltg[j]][jj] >= 0 )
									{
										global_polynomial_vector( lbase[ltg[j]][jj], dim, expr );
										if( b_use_singular )
											x2 = punity_eval_local_poly_delta( pu, ltg[j], expr, qx, i_sing_order );
										else
											x2 = punity_eval_local_poly( pu, ltg[j], expr, qx );
									}
									else
									{
										/* Deal with case in which lbase[ltg[j]][jj] < 0 */
										idx = 1 - idx;
										
									}

									/* Do the evaluation */
									if( !b_load_overl_mat )
										sum += qw * x1 * x2;
									if( !b_load_stiff_mat && b_use_external_potential )
										rum += qw * x1 * x2 * nuclei_potential( nuc, qx );
								}
								if( !b_load_stiff_mat )
								{
									y = 0.0;
									for(k=0;k<dim;k++)
									{
										for(m=0;m<dim;m++)
										{
											if( m == k )
												drv[m] = 1;
											else
												drv[m] = 0;
										}
										if( lbase[ltg[i]][ii] >= 0 )
										{
											global_polynomial_vector( lbase[ltg[i]][ii], dim, expl );
											if( b_use_singular )
												x1 = punity_term_deriv_local_poly_delta( pu, ltg[i], expl, drv, qx, i_sing_order, nb, nnb );
											else
												x1 = punity_term_deriv_local_poly( pu, ltg[i], expl, drv, qx, nb, nnb );
										}
										else
										{
											/* Deal with other case */
											idx = 1 - idx;
											
										}
										if( lbase[ltg[j]][jj] >= 0 )
										{
											global_polynomial_vector( lbase[ltg[j]][jj], dim, expr );
											if( b_use_singular )
												x2 = punity_term_deriv_local_poly_delta( pu, ltg[j], expr, drv, qx, i_sing_order, nb, nnb );
											else
												x2 = punity_term_deriv_local_poly( pu, ltg[j], expr, drv, qx, nb, nnb );
										}
										else
										{
											/* Deal with other case */
											idx = 1 - idx;
											
										}
										y += 0.5 * x1 * x2;
									}
									tum += qw * y;
								}
								if( nnb > 0 )
									free( nb );
							}
						}
						while( arraynext( (long) dim, size, index ) != -1 );

						/* Now put sum in the i,j entry in the sparse matrx of inner product entries */
						if( p > cc )
						{
							cc += c_spmat_inc;
							if( !b_load_stiff_mat )
							{
								A = (double*) realloc( A, cc * sizeof(double) );
								ja = (long*) realloc( ja, cc * sizeof(long) );
							}
							if( !b_load_overl_mat )
							{
								B = (double*) realloc( B, cc * sizeof(double) );
								jb = (long*) realloc( jb, cc * sizeof(long) );
							}
						}
						if( !b_load_stiff_mat )
						{
							A[p] = tum;
							if( b_use_external_potential )
								A[p] += rum; /* Only add this term in if reading in an external potential */
							ja[p] = qq;
							if( ia[pp] == -1 )
								ia[pp] = p;
						}
						if( !b_load_overl_mat )
						{
							B[p] = sum;
							jb[p] = qq;
							if( ib[pp] == -1 )
								ib[pp] = p;
						}

						/* Step to the next entry */
						p++;
					}
					qq++;
				}
			}
			pp++;
		}
		if( npts >= 20 )
			parse_print_back( stderr, i / ( npts / 20 ) + 1 + 4 );
	}
	if( !b_load_stiff_mat )
		ia[mdim] = p;
	if( !b_load_overl_mat )
		ib[mdim] = p;
	fprintf( stderr, "\n\n" );

	/* Indicate that matrices are both built */
	if( !b_load_stiff_mat )
		*b_have_stiff_mat = 1;
	if( !b_load_overl_mat )
		*b_have_overl_mat = 1;

	/* Pass the variables back */
	*iap = ia; *ibp = ib;
	*jap = ja; *jbp = jb;
	*Ap = A; *Bp = B;
}

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

int main()
{
	
}

